#include<stdio.h>
#include<math.h>
#include<hip/hip_runtime.h>
#define TWW 32 // Setting TileWidth

/*----------Kernel Function------------*/
__global__ void matMul_Soft_prefetch(double *d_a, double *d_b, double *d_c, int M, int N, int K){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col= blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ double ds_a[TWW][TWW];
	__shared__ double ds_b[TWW][TWW];

	double cval=0.0;
	
	// Loading data from Global meomory to register
	register double reg_1 =d_a[row*N+threadIdx.x];
	register double reg_2 =d_b[threadIdx.y*K+col];

	//For loopstarts from 1 (instead of 0 like in the tiled matrix multiplication) 
	//because we have already loaded data of phase 0 in register
	for(int t=1;t<N/TWW;t++){
		
		//Load data of current phase into shared memory 
		ds_a[threadIdx.y][threadIdx.x]=reg_1;
		ds_b[threadIdx.y][threadIdx.x]=reg_2;

		__syncthreads();
		
		//Starting fetching data for next phase in register
               	reg_1 = d_a[row*N+t*TWW+threadIdx.x];
		reg_2 = d_b[(t*TWW+threadIdx.y)*K+col];
		
		for(int k=0;k<TWW;k++){
			cval+=ds_a[threadIdx.y][k]*ds_b[k][threadIdx.x];
		}
		
		__syncthreads();

	}
	
	//For last phase data loading in shared memory
	ds_a[threadIdx.y][threadIdx.x]=reg_1;
	ds_b[threadIdx.y][threadIdx.x]=reg_2;

	__syncthreads();		
		
	for(int k=0;k<TWW;k++){
        	cval+=ds_a[threadIdx.y][k]*ds_b[k][threadIdx.x];
        }
	__syncthreads();

	d_c[row*K + col]=cval;

}
/*------------------------------*/

int main(int argc, char const *argv[]) {
	int N=800;
	int M=N,K=N;
	double h_a[M][N],h_b[N][K],h_c[M][K];
	double *d_a,*d_b,*d_c;
	hipEvent_t start,stop;
	float ms;
	for(int i=0;i<M;i++){
		for(int j=0;j<N;j++){
			h_a[i][j]=rand()%100;
		}
	}
	for(int i=0;i<N;i++){
		for(int j=0;j<K;j++){
			h_b[i][j]=rand()%100;
		}
	}

	/*printf("\n A Matrix\n" );
	for(int i=0;i<M;i++){
		for(int j=0;j<N;j++){
			printf("%0.1f ",h_a[i][j] );
		}
		printf("\n" );
	}
	printf("\n B Matrix\n" );
	for(int i=0;i<N;i++){
		for(int j=0;j<K;j++){
			printf("%0.1f ",h_b[i][j] );
		}
		printf("\n" );
	}*/

	// taking block diamension as TWW X TWW
	dim3 dimBlock(TWW,TWW);
	dim3 dimGrid(K/TWW,M/TWW );


	// allocating device memory
	hipMalloc(&d_a, M*N*sizeof(double));
	hipMalloc(&d_b, N*K*sizeof(double));
	hipMalloc(&d_c, M*K*sizeof(double));


	// copying data in device memory
	hipMemcpy( d_a, h_a, M*N*sizeof(double), hipMemcpyHostToDevice   );
	hipMemcpy( d_b, h_b, N*K*sizeof(double), hipMemcpyHostToDevice   );

	//Creating timestamp event
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	//Recording Kernel start time
	hipEventRecord(start, 0);	
	
	//calling kernel function
	matMul_Soft_prefetch<<<dimGrid,dimBlock>>>(d_a,d_b,d_c,M,N,K);
	
	//Recording Kernel stop time
	hipEventRecord(stop, 0);
	
	hipMemcpy(h_c, d_c, M*K*sizeof(double), hipMemcpyDeviceToHost  );

	hipEventSynchronize(stop);
	hipEventElapsedTime(&ms, start, stop);

	printf("\nTime:%f ", ms);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	/*printf("\n Multiplication of A and B Matrix using Tiling:\n" );
	for(int i=0;i<M;i++){
		for(int j=0;j<K;j++){
			printf("%0.1f ",h_c[i][j] );
		}
		printf("\n" );
	}*/
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}


