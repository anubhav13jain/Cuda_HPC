#include<stdio.h>
#include<math.h>
#include<hip/hip_runtime.h>
#define TWW 32		// Setting TileWidth


/*----------Kernel Function------------*/
__global__ void matMul(double *d_a,double *d_b,double *d_c, int M, int N, int K){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col= blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ double ds_a[TWW][TWW];
	__shared__ double ds_b[TWW][TWW];

	double cval=0.0;
	
	for(int t=0;t<N/TWW;t++){

		// Loading data from Global meomory to Shared memory
		ds_a[threadIdx.y][threadIdx.x]=d_a[row*N+t*TWW+threadIdx.x];
		ds_b[threadIdx.y][threadIdx.x]=d_b[(t*TWW+threadIdx.y)*K+col];

		__syncthreads();

		for(int k=0;k<TWW;k++){
			cval+=ds_a[threadIdx.y][k]*ds_b[k][threadIdx.x];
		}
		__syncthreads();
	}
	d_c[row*K + col]=cval;

}
/*------------------------------*/

int main(int argc, char const *argv[]) {
	/*Matrix A size = M X N and Matrix B size = N X K*/
	
	int N=800, M=N,K=N;
	double h_a[M][N],h_b[N][K],h_c[M][K];
	double *d_a,*d_b,*d_c;
	hipEvent_t start,stop;
	float ms;

	//Generatig matrix
	for(int i=0;i<M;i++){
		for(int j=0;j<N;j++){
			h_a[i][j]=rand()%100;
		}
	}
	for(int i=0;i<N;i++){
		for(int j=0;j<K;j++){
			h_b[i][j]=rand()%100;
		}
	}

	/*printf("\n A Matrix\n" );
	for(int i=0;i<M;i++){
		for(int j=0;j<N;j++){
			printf("%0.1f ",h_a[i][j] );
		}
		printf("\n" );
	}
	printf("\n B Matrix\n" );
	for(int i=0;i<N;i++){
		for(int j=0;j<K;j++){
			printf("%0.1f ",h_b[i][j] );
		}
		printf("\n" );
	}*/

	// taking block diamension as TWW X TWW
	dim3 dimBlock(TWW,TWW);
	dim3 dimGrid(K/TWW,M/TWW );


	// allocating device memory
	hipMalloc(&d_a, M*N*sizeof(double));
	hipMalloc(&d_b, N*K*sizeof(double));
	hipMalloc(&d_c, M*K*sizeof(double));


	// copying data in device memory
	hipMemcpy( d_a, h_a, M*N*sizeof(double), hipMemcpyHostToDevice   );
	hipMemcpy( d_b, h_b, N*K*sizeof(double), hipMemcpyHostToDevice   );

	//Creating timestamp event
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//Recording Kernel start time
	hipEventRecord(start, 0);

	//calling kernel function
	matMul<<<dimGrid,dimBlock>>>(d_a,d_b,d_c,M,N,K);

	//Recording Kernel stop time
	hipEventRecord(stop, 0);
	hipMemcpy(h_c, d_c, M*K*sizeof(double), hipMemcpyDeviceToHost  );

	hipEventSynchronize(stop);
	hipEventElapsedTime(&ms, start, stop);
	
	printf("\nTime:%f ",ms );
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipMemcpy(h_c, d_c, M*K*sizeof(double), hipMemcpyDeviceToHost  );

	/*printf("\n Multiplication of A and B Matrix using Tiling:\n" );
	for(int i=0;i<M;i++){
		for(int j=0;j<K;j++){
			printf("%0.1f ",h_c[i][j] );
		}
		printf("\n" );
	}*/
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}


