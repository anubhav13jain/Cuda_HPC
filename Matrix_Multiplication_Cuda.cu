#include<stdio.h>
#include<math.h>
#include<hip/hip_runtime.h>

__global__ void matMul(int *d_a,int *d_b,int *d_c, int M, int N, int K){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col= blockIdx.x * blockDim.x + threadIdx.x;

	if(row<M && col<K){
		int sum=d_a[row*N]*d_b[col];
		for(int k=1;k<N;k++){
			sum+=d_a[row*N+k]*d_b[k*K+col];
		}
		d_c[row*K + col]=sum;
	}
}

int main(int argc, char const *argv[]) {
	int N=8,M=10,K=9,TW=4;
	int h_a[M][N],h_b[N][K],h_c[M][K];
	int *d_a,*d_b,*d_c;

	for(int i=0;i<M;i++){
		for(int j=0;j<N;j++){
			h_a[i][j]=rand()%100;
		}
	}
	for(int i=0;i<N;i++){
		for(int j=0;j<K;j++){
			h_b[i][j]=rand()%100;
		}
	}

	printf("\n A Matrix\n" );
	for(int i=0;i<M;i++){
		for(int j=0;j<N;j++){
			printf("%d ",h_a[i][j] );
		}
		printf("\n" );
	}
	printf("\n B Matrix\n" );
	for(int i=0;i<N;i++){
		for(int j=0;j<K;j++){
			printf("%d ",h_b[i][j] );
		}
		printf("\n" );
	}

	// taking block diamension as M X M
	dim3 dimBlock(TW,TW);
	dim3 dimGrid((int)((M-1)/TW)+1,(int)((K-1)/TW)+1,1 );

	// allocating device memory
	hipMalloc(&d_a, M*N*sizeof(int));
	hipMalloc(&d_b, N*K*sizeof(int));
	hipMalloc(&d_c, M*K*sizeof(int));


	// copying data in device memory
	hipMemcpy( d_a, h_a, M*N*sizeof(int), hipMemcpyHostToDevice   );
	hipMemcpy( d_b, h_b, N*K*sizeof(int), hipMemcpyHostToDevice   );

	//calling kernel function

	matMul<<<dimGrid,dimBlock>>>(d_a,d_b,d_c,M,N,K);

	hipMemcpy(h_c, d_c, M*K*sizeof(int), hipMemcpyDeviceToHost  );

	printf("\n Multiplication of A and B Matrix:\n" );
	for(int i=0;i<M;i++){
		for(int j=0;j<K;j++){
			printf("%d ",h_c[i][j] );
		}
		printf("\n" );
	}
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}
